#include "hip/hip_runtime.h"
/**
 * @author Paul Arlt, Pascal Miliczek
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/sparse/csr_svm_kernel.cuh"

#include "plssvm/backends/CUDA/detail/atomics.cuh"  // atomicAdd
#include "plssvm/constants.hpp"                     // plssvm::INTERNAL_BLOCK_SIZE, plssvm::kernel_index_type

// UNTESTED
namespace plssvm::cuda {

template <typename real_type>
__global__ void device_kernel_linear(const real_type *q, real_type *ret, const real_type *d, const size_t *col_ids, const size_t *row_ids, const real_type *values, const real_type QA_cost, const real_type cost, const kernel_index_type nnz, const kernel_index_type height, const real_type add) {
    kernel_index_type i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;
    kernel_index_type j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;

    if (i < j) {
        return;
    }

    kernel_index_type row_1_start_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_2_start_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_1_end_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_2_end_indices[INTERNAL_BLOCK_SIZE] = { 0 };

    #pragma unroll INTERNAL_BLOCK_SIZE
    for (kernel_index_type block_index = 0; block_index < INTERNAL_BLOCK_SIZE; ++block_index) {
        row_1_start_indices[block_index] = row_offsets[i + blockindex];
        row_2_start_indices[block_index] = row_offsets[j + blockindex];

        row_1_end_indices[block_index] = nnz;
        if(i + block_index < height - 1){
            row_1_end_indices[block_index] = row_offsets[i + blockindex + 1];
        }

        row_2_end_indices[block_index] = nnz;
        if(j + block_index < height - 1){
            row_2_end_indices[block_index] = row_offsets[j + blockindex + 1];
        }
    }

    #pragma unroll INTERNAL_BLOCK_SIZE
    for (kernel_index_type y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
        real_type ret_jy = 0.0;
        #pragma unroll INTERNAL_BLOCK_SIZE
        for (kernel_index_type x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            kernel_index_type row_1_index = row_1_start_indices[x];
            kernel_index_type row_2_index = row_2_start_indices[y];
            real_type matr_ix_jy = 0.0;

            // multiply rows
            while (row_1_index < row_1_end_indices[x] && row_2_index < row_2_end_indices[y]) {
                if (col_ids[row_1_index] == col_ids[row_2_index]) {
                    matr_ix_jy += values[row_1_index] * values[row_2_index];
                    row_1_index++;
                    row_2_index++;
                } else if (col_ids[row_1_index] < col_ids[row_2_index]) {
                    row_1_index++;
                } else {
                    row_2_index++;
                }
            }

            real_type temp = (matr_ix_jy + QA_cost - q[i + x] - q[j + y]) * add;
            if (i + x > j + y) {
                atomicAdd(&ret[i + x], temp * d[j + y]);
                ret_jy += temp * d[i + x];
            } else if (i + x == j + y) {
                ret_jy += (temp + cost * add) * d[i + x];
            }
        }
        atomicAdd(&ret[j + y], ret_jy);
    }
}
template __global__ void device_kernel_linear(const float *, float *, const float *, const size_t *, const size_t *, const float *, const float, const float, const kernel_index_type, const kernel_index_type, const float);
template __global__ void device_kernel_linear(const double *, double *, const double *, const size_t *, const size_t *, const double *, const double, const double, const kernel_index_type, const kernel_index_type, const double);

template <typename real_type>
__global__ void device_kernel_poly(const real_type *q, real_type *ret, const real_type *d, const size_t *col_ids, const size_t *row_ids, const real_type *values, const real_type QA_cost, const real_type cost, const kernel_index_type nnz, const kernel_index_type height, const real_type add, const int degree, const real_type gamma, const real_type coef0) {
    kernel_index_type i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;
    kernel_index_type j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;

    if (i < j) {
        return;
    }

    kernel_index_type row_1_start_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_2_start_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_1_end_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_2_end_indices[INTERNAL_BLOCK_SIZE] = { 0 };

    #pragma unroll INTERNAL_BLOCK_SIZE
    for (kernel_index_type block_index = 0; block_index < INTERNAL_BLOCK_SIZE; ++block_index) {
        row_1_start_indices[block_index] = row_offsets[i + blockindex];
        row_2_start_indices[block_index] = row_offsets[j + blockindex];

        row_1_end_indices[block_index] = nnz;
        if(i + block_index < height - 1){
            row_1_end_indices[block_index] = row_offsets[i + blockindex + 1];
        }

        row_2_end_indices[block_index] = nnz;
        if(j + block_index < height - 1){
            row_2_end_indices[block_index] = row_offsets[j + blockindex + 1];
        }
    }

    #pragma unroll INTERNAL_BLOCK_SIZE
    for (kernel_index_type y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
        real_type ret_jy = 0.0;
        #pragma unroll INTERNAL_BLOCK_SIZE
        for (kernel_index_type x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            kernel_index_type row_1_index = row_1_start_indices[x];
            kernel_index_type row_2_index = row_2_start_indices[y];
            real_type matr_ix_jy = 0.0;

            // multiply rows
            while (row_1_index < row_1_end_indices[x] && row_2_index < row_2_end_indices[y]) {
                if (col_ids[row_1_index] == col_ids[row_2_index]) {
                    matr_ix_jy += values[row_1_index] * values[row_2_index];
                    row_1_index++;
                    row_2_index++;
                } else if (col_ids[row_1_index] < col_ids[row_2_index]) {
                    row_1_index++;
                } else {
                    row_2_index++;
                }
            }

            real_type temp = (pow(gamma * matr_ix_jy + coef0, degree) + QA_cost - q[i + x] - q[j + y]) * add;
            if (i + x > j + y) {
                atomicAdd(&ret[i + x], temp * d[j + y]);
                ret_jy += temp * d[i + x];
            } else if (i + x == j + y) {
                ret_jy += (temp + cost * add) * d[i + x];
            }
        }
        atomicAdd(&ret[j + y], ret_jy);
    }
}
template __global__ void device_kernel_poly(const float *, float *, const float *, const size_t *, const size_t *, const float *, const float, const float, const kernel_index_type, const kernel_index_type, const float, const int, const float, const float);
template __global__ void device_kernel_poly(const double *, double *, const double *, const size_t *, const size_t *, const double *, const double, const double, const kernel_index_type, const kernel_index_type, const double, const int, const double, const double);

template <typename real_type>
__global__ void device_kernel_radial(const real_type *q, real_type *ret, const real_type *d, const size_t *col_ids, const size_t *row_ids, const real_type *values, const real_type QA_cost, const real_type cost, const kernel_index_type nnz, const kernel_index_type height, const real_type add, const real_type gamma) {
    kernel_index_type i = (blockIdx.x * blockDim.x + threadIdx.x) * INTERNAL_BLOCK_SIZE;
    kernel_index_type j = (blockIdx.y * blockDim.y + threadIdx.y) * INTERNAL_BLOCK_SIZE;

    if (i < j) {
        return;
    }

    kernel_index_type row_1_start_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_2_start_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_1_end_indices[INTERNAL_BLOCK_SIZE] = { 0 };
    kernel_index_type row_2_end_indices[INTERNAL_BLOCK_SIZE] = { 0 };

    #pragma unroll INTERNAL_BLOCK_SIZE
    for (kernel_index_type block_index = 0; block_index < INTERNAL_BLOCK_SIZE; ++block_index) {
        row_1_start_indices[block_index] = row_offsets[i + blockindex];
        row_2_start_indices[block_index] = row_offsets[j + blockindex];

        row_1_end_indices[block_index] = nnz;
        if(i + block_index < height - 1){
            row_1_end_indices[block_index] = row_offsets[i + blockindex + 1];
        }

        row_2_end_indices[block_index] = nnz;
        if(j + block_index < height - 1){
            row_2_end_indices[block_index] = row_offsets[j + blockindex + 1];
        }
    }

    for (kernel_index_type y = 0; y < INTERNAL_BLOCK_SIZE; ++y) {
        real_type ret_jy = 0.0;
        for (kernel_index_type x = 0; x < INTERNAL_BLOCK_SIZE; ++x) {
            kernel_index_type row_1_index = row_1_start_indices[x];
            kernel_index_type row_2_index = row_2_start_indices[y];
            real_type matr_ix_jy = 0.0;

            // calc sq. e. dist
            while (row_1_index < row_1_end_indices[x] && row_2_index < row_2_end_indices[y]) {
                if (col_ids[row_1_index] == col_ids[row_2_index]) {
                    matr_ix_jy += (values[row_1_index] - values[row_2_index]) * (values[row_1_index] - values[row_2_index]);
                    row_1_index++;
                    row_2_index++;
                } else if (col_ids[row_1_index] < col_ids[row_2_index]) {
                    matr_ix_jy += values[row_1_index] * values[row_1_index];
                    row_1_index++;
                } else {
                    matr_ix_jy += values[row_2_index] * values[row_2_index];
                    row_2_index++;
                }
            }
            
            for (;row_1_index < row_1_end_indices[x]; ++row_1_index) {
                matr_ix_jy += values[row_1_index] * values[row_1_index];
            }

            for (;row_2_index < row_2_end_indices[y]; ++row_2_index) {
                matr_ix_jy += values[row_2_index] * values[row_2_index];
            }
            
            float temp = (exp(-gamma * matr_ix_jy) + QA_cost - q[i + x] - q[j + y]) * add;
            if (i + x > j + y) {
                atomicAdd(&ret[i + x], temp * d[j + y]);
                ret_jy += temp * d[i + x];
            } else if (i + x == j + y) {
                ret_jy += (temp + cost * add) * d[i + x];
            }
        }
        atomicAdd(&ret[j + y], ret_jy);
    }
}
template __global__ void device_kernel_radial(const float *, float *, const float *, const size_t *, const size_t *, const float *, const float, const float, const kernel_index_type, const kernel_index_type, const float, const float,);
template __global__ void device_kernel_radial(const double *, double *, const double *, const size_t *, const size_t *, const double *, const double, const double, const kernel_index_type, const kernel_index_type, const double, const double);

}  // namespace plssvm::cuda
