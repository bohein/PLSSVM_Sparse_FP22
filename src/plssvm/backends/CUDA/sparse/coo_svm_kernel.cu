#include "hip/hip_runtime.h"
/**
 * @author Paul Arlt
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/sparse/coo_svm_kernel.cuh"

namespace plssvm::cuda {

template <typename real_type>
__global__ void device_kernel_linear(const real_type *q, real_type *ret, const real_type *d, const size_t *col_ids, const size_t *row_ids, const real_type *values, const real_type QA_cost, const real_type cost, const real_type add) {
    kernel_index_type i = blockIdx.x * blockDim.x + threadIdx.x;
    kernel_index_type j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < j) {
        return;
    }

    kernel_index_type row_id_i = static_cast<kernel_index_type>(row_ids[i]);
    kernel_index_type row_id_j = static_cast<kernel_index_type>(row_ids[j]);

    const real_type temp = (values[i] * values[j] + QA_cost - q[row_id_i] - q[row_id_j]) * add;
    if (i == j) {
        ret[row_id_i] += (temp + cost * add) * d[row_id_i];
    } else {
        ret[row_id_i] += temp * d[row_id_j];
        ret[row_id_j] += temp * d[row_id_i];
    }
}
template __global__ void device_kernel_linear(const float *, float *, const float *, const size_t *, const size_t *, const float *, const float, const float, const float);
template __global__ void device_kernel_linear(const double *, double *, const double *, const size_t *, const size_t *, const double *, const double, const double, const double);

template <typename real_type>
__global__ void device_kernel_poly(const real_type *q, real_type *ret, const real_type *d, const size_t *col_ids, const size_t *row_ids, const real_type *values, const real_type QA_cost, const real_type cost, const real_type add, const int degree, const real_type gamma, const real_type coef0) {
    kernel_index_type i = blockIdx.x * blockDim.x + threadIdx.x;
    kernel_index_type j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < j) {
        return;
    }

    kernel_index_type row_id_i = static_cast<kernel_index_type>(row_ids[i]);
    kernel_index_type row_id_j = static_cast<kernel_index_type>(row_ids[j]);

    const real_type temp = (pow(gamma * values[i] * values[j] + coef0, degree) + QA_cost - q[row_id_i] - q[row_id_j]) * add;
    if (i == j) {
        ret[row_id_i] += (temp + cost * add) * d[row_id_i];
    } else {
        ret[row_id_i] += temp * d[row_id_j];
        ret[row_id_j] += temp * d[row_id_i];
    }
}
template __global__ void device_kernel_poly(const float *, float *, const float *, const size_t *, const size_t *, const float *, const float, const float, const float, const int, const float, const float);
template __global__ void device_kernel_poly(const double *, double *, const double *, const size_t *, const size_t *, const double *, const double, const double, const double, const int, const double, const double);

template <typename real_type>
__global__ void device_kernel_radial(const real_type *q, real_type *ret, const real_type *d, const size_t *col_ids, const size_t *row_ids, const real_type *values, const real_type QA_cost, const real_type cost, const real_type add, const real_type gamma) {
    kernel_index_type i = blockIdx.x * blockDim.x + threadIdx.x;
    kernel_index_type j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < j) {
        return;
    }

    kernel_index_type row_id_i = static_cast<kernel_index_type>(row_ids[i]);
    kernel_index_type row_id_j = static_cast<kernel_index_type>(row_ids[j]);

    const real_type temp = (exp(-gamma * pow(values[i] - values[j], 2)) + QA_cost - q[row_id_i] - q[row_id_j]) * add;
    if (i == j) {
        ret[row_id_i] += (temp + cost * add) * d[row_id_i];
    } else {
        ret[row_id_i] += temp * d[row_id_j];
        ret[row_id_j] += temp * d[row_id_i];
    }
}
template __global__ void device_kernel_radial(const float *, float *, const float *, const size_t *, const size_t *, const float *, const float, const float, const float, const float);
template __global__ void device_kernel_radial(const double *, double *, const double *, const size_t *, const size_t *, const double *, const double, const double, const double, const double);

}  // namespace plssvm::cuda
