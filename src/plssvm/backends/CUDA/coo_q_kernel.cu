#include "hip/hip_runtime.h"
/**
 * @author Paul Arlt
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/coo_q_kernel.cuh"

#include "plssvm/constants.hpp" 

// UNTESTED
namespace plssvm::cuda {

template <typename real_type>
__global__ void device_kernel_q_linear(real_type *q, const kernel_index_type *col_ids, const kernel_index_type *row_ids, const real_type *values, const kernel_index_type last_row_begin, const kernel_index_type num_cols) {
    
    const kernel_index_type row_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    kernel_index_type search_index = row_index * last_row_begin / gridDim.x;
    real_type temp{ 0.0 };

    if (row_ids[search_index] < row_index) {
        for (; search_index < last_row_begin && row_ids[search_index] < row_index; ++search_index);
    } else {
        for (; search_index >= 0 && row_ids[search_index] >= row_index; --search_index);
        search_index++;
    }

    for (kernel_index_type last_row_index = last_row_begin; last_row_index < num_cols; ++last_row_index) {
        for (; search_index < last_row_begin && col_ids[search_index] < col_ids[last_row_index]; ++search_index);
        if (row_ids[search_index] != row_index) {
            break;
        }
        if (col_ids[search_index] == col_ids[last_row_index]) {
            temp += values[search_index] * values[last_row_index];
        }
    }
    
    q[row_index] = temp;
}
template __global__ void device_kernel_q_linear(float *, const kernel_index_type *, const kernel_index_type *, const float *, const kernel_index_type, const kernel_index_type);
template __global__ void device_kernel_q_linear(double *, const kernel_index_type *, const kernel_index_type *, const double *, const kernel_index_type, const kernel_index_type);

template <typename real_type>
__global__ void device_kernel_q_poly(real_type *q, const kernel_index_type *col_ids, const kernel_index_type *row_ids, const real_type *values, const kernel_index_type last_row_begin, const kernel_index_type num_cols, const int degree, const real_type gamma, const real_type coef0) {
    const kernel_index_type row_index = blockIdx.x * blockDim.x + threadIdx.x;
    kernel_index_type search_index = row_index * last_row_begin / gridDim.x;
    real_type temp{ 0.0 };

    if (row_ids[search_index] < row_index) {
        for (; search_index < last_row_begin && row_ids[search_index] < row_index; ++search_index);
    } else {
        for (; search_index >= 0 && row_ids[search_index] >= row_index; --search_index);
        search_index++;
    }

    for (kernel_index_type last_row_index = last_row_begin; last_row_index < num_cols; ++last_row_index) {
        for (; search_index < last_row_begin && col_ids[search_index] < col_ids[last_row_index]; ++search_index);
        if (row_ids[search_index] != row_index) {
            break;
        }
        if (col_ids[search_index] == col_ids[last_row_index]) {
            temp += values[search_index] * values[last_row_index];
        }
    }
    
    q[row_index] = pow(gamma * temp + coef0, degree);
}
template __global__ void device_kernel_q_poly(float *, const kernel_index_type *, const kernel_index_type *, const float *, const kernel_index_type, const kernel_index_type, const int, const float, const float);
template __global__ void device_kernel_q_poly(double *, const kernel_index_type *, const kernel_index_type *, const double *, const kernel_index_type,const kernel_index_type, const int, const double, const double);

template <typename real_type>
__global__ void device_kernel_q_radial(real_type *q, const kernel_index_type *col_ids, const kernel_index_type *row_ids, const real_type *values, const kernel_index_type last_row_begin,const kernel_index_type num_cols, const real_type gamma) {
    const kernel_index_type row_index = blockIdx.x * blockDim.x + threadIdx.x;
    kernel_index_type search_index = row_index * last_row_begin / gridDim.x;
    real_type temp{ 0.0 };

    if (row_ids[search_index] < row_index) {
        for (; search_index < last_row_begin && row_ids[search_index] < row_index; ++search_index);
    } else {
        for (; search_index >= 0 && row_ids[search_index] >= row_index; --search_index);
        search_index++;
    }

    for (kernel_index_type last_row_index = last_row_begin; last_row_index < num_cols; ++last_row_index) {
        for (; search_index < last_row_begin && col_ids[search_index] < col_ids[last_row_index]; ++search_index);
        if (row_ids[search_index] != row_index) {
            break;
        }
        if (col_ids[search_index] == col_ids[last_row_index]) {
            temp += pow(values[search_index] - values[last_row_index], 2);
        }
    }
    
    q[row_index] = exp(-gamma * temp);
}
template __global__ void device_kernel_q_radial(float *, const kernel_index_type *, const kernel_index_type *, const float *, const kernel_index_type,const kernel_index_type, const float);
template __global__ void device_kernel_q_radial(double *, const kernel_index_type *, const kernel_index_type *, const double *, const kernel_index_type,const kernel_index_type, const double);
}  // namespace plssvm::cuda
