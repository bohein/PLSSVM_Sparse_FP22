#include "hip/hip_runtime.h"
/**
 * @file
 * @author Tim Schmidt
 * @author Pascal Miliczek
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 *
 * @brief Defines the base class for benchmarks reagrding svm-kernel functions.
 */

#include "plssvm/benchmarks/benchmark_svm_kernel_cuda.cuh"

#include "plssvm/backends/CUDA/q_kernel.cuh"
#include "plssvm/backends/CUDA/sparse/coo_q_kernel.cuh"
#include "plssvm/backends/CUDA/sparse/csr_q_kernel.cuh"

#include "plssvm/backends/CUDA/svm_kernel.cuh"
#include "plssvm/backends/CUDA/sparse/coo_svm_kernel.cuh"
#include "plssvm/backends/CUDA/sparse/csr_svm_kernel.cuh"

#include <numeric>
#include <iostream>

namespace plssvm::benchmarks {

benchmark_svm_kernel_cuda::benchmark_svm_kernel_cuda() : benchmark{"SVM-Kernels (CUDA)"} {}

void benchmark_svm_kernel_cuda::run() {
    using real_type = double;

    //evaluate_dataset("tiny (~150)", DATASET_TINY);
    //evaluate_dataset("small (~5000)", DATASET_SMALL);
    //evaluate_dataset("medium (~50000)", DATASET_MEDIUM);
    //evaluate_dataset("large (~250000)", DATASET_LARGE);
}

void benchmark_svm_kernel_cuda::evaluate_dataset(const dataset& ds) {
    using real_type = double;

    std::chrono::time_point start_time = std::chrono::high_resolution_clock::now();
    std::chrono::time_point end_time = std::chrono::high_resolution_clock::now();

    plssvm::parameter<real_type> params;
    std::vector<real_type> q; // q-Vector
    std::vector<real_type> q_d; // q-Vector on device
    real_type QA_cost;
    real_type QA_cost_d;
    std::vector<real_type> ret; // result Vector
    std::vector<real_type> ret_d; // result Vector on device
    std::vector<real_type> d; // ""right-hand side of the equation"
    std::vector<real_type> d_d; // ""right-hand side of the equation" on device

    real_type cost_d;
    real_type add_d;
    int degree_d;
    real_type gamma_d;
    real_type coef0_d;
    

    hipMalloc((void*)&cost_d, sizeof(real_type));
    hipMemcpy(cost_d, cost, sizeof(real_type), hipMemcpyHostToDevice);
    
    hipMalloc((void*)&add_d, sizeof(real_type));
    hipMemcpy(add_d, add, sizeof(real_type), hipMemcpyHostToDevice);
    
    hipMalloc((void*)&degree_d, sizeof(int));
    hipMemcpy(degree_d, degree, sizeof(real_type), hipMemcpyHostToDevice);
    
    hipMalloc((void*)&gamma_d, sizeof(real_type));
    hipMemcpy(gamma_d, gamma, sizeof(real_type), hipMemcpyHostToDevice);
    
    hipMalloc((void*)&coef0_d, sizeof(real_type));
    hipMemcpy(coef0_d, coef0, sizeof(real_type), hipMemcpyHostToDevice);

    std::vector<std::vector<real_type>> data_dense;
    std::vector<real_type> data_dense_d;

    plssvm::openmp::coo<real_type> data_coo{};
    std::vector<real_type> values_coo_d;
    std::vector<size_t> row_coo_d;
    std::vector<size_t> col_coo_d;
    size_t nnz_coo_d;
    size_t last_row_begin_coo_d;
    

    plssvm::openmp::csr<real_type> data_csr{};
    std::vector<real_type> values_csr_d;
    std::vector<size_t> row_csr_d;
    std::vector<size_t> col_csr_d;
    size_t nnz_csr_d;
    size_t height_csr_d;

    auto data_ptr_dense = std::make_shared<const std::vector<std::vector<real_type>>>(std::move(data_dense));
    auto data_ptr_coo = std::make_shared<const plssvm::openmp::coo<real_type>>(std::move(data_coo));
    auto data_ptr_csr = std::make_shared<const plssvm::openmp::csr<real_type>>(std::move(data_csr));

    // dense
    std::vector<ns> raw_runtimes_dense_linear;
    std::vector<ns> raw_runtimes_dense_poly;
    std::vector<ns> raw_runtimes_dense_radial;
    params.parse_libsvm_file(ds.path, data_ptr_dense);
    auto data_ptr_dense_1D = std::make_shared<const std::vector<real_type>>(plssvm::csvm<real_type>::transform_data(data_ptr_dense.get(), 0, ((*data_ptr_dense.get())[0].size()) * (data_ptr_dense.get() -> size()))); //padding----------------------

    auto data_dense_last = std::make_shared<const std::vector<real_type>>((*data_ptr_dense.get())[data_ptr_dense.get() -> size() - 1]);
    std::vector<real_type> data_dense_d;
    std::vector<real_type> data_dense_last_d;
    int num_rows_d;
    int num_cols_d;
    int id_d;
    
    size_t boundary_size = static_cast<std::size_t>(THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE);
    size_t num_rows_exc_last;

    size_t boundary_size = static_cast<std::size_t>(THREAD_BLOCK_SIZE * INTERNAL_BLOCK_SIZE);
    size_t num_rows_exc_last = data_ptr_dense.get() -> size() - 1;

    const plssvm::detail::execution_range range_q({ static_cast<std::size_t>(std::ceil(static_cast<real_type>(num_rows_exc_last) / static_cast<real_type>(THREAD_BLOCK_SIZE))) },
                                            { std::min<std::size_t>(THREAD_BLOCK_SIZE, num_rows_exc_last) });
    dim3 grid_q(range_q.grid[0], range_q.grid[1], range_q.grid[2]);
    dim3 block_q(range_q.block[0], range_q.block[1], range_q.block[2]); 

    const auto grid = static_cast<std::size_t>(std::ceil(static_cast<real_type>(num_rows_exc_last) / static_cast<real_type>(boundary_size)));
    const plssvm::detail::execution_range range_svm({ grid, grid }, { THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE });

    dim3 grid_svm(range_svm.grid[0], range_svm.grid[1], range_svm.grid[2]);
    dim3 block_svm(range_svm.block[0], range_svm.block[1], range_svm.block[2]); 

    

    for(size_t i = 0; i < cycles; i++) {
        hipMalloc((void**)&q_d, sizeof(real_type)*(data_ptr_dense -> size()));
        hipMalloc((void**)&ret_d, sizeof(real_type)*(data_ptr_dense -> size()));
        hipMalloc((void**)&d_d, sizeof(real_type)*(data_ptr_dense -> size()));
        hipMalloc((void**)&QA_cost_d, sizeof(real_type));
        hipMalloc((void**)&id_d, sizeof(int));

        hipMalloc((void**)&data_dense_d, sizeof(real_type)*(data_ptr_dense_1D.get() -> size()));
        hipMalloc((void**)&num_rows_d, sizeof(int));
        hipMalloc((void**)&num_cols_d, sizeof(int));
        hipMalloc((void**)&data_dense_last_d, sizeof(real_type)*(*data_ptr_dense_1D.get())[0].size());


        hipMemcpy(data_dense_d, data_ptr_dense_1D.get(), sizeof(real_type)*(data_ptr_dense_1D.get() -> size()));
        hipMemcpy(num_rows_d, data_ptr_dense.get() -> size(), sizeof(int));
        hipMemcpy(num_cols_d, (*data_ptr_dense.get())[0].size(), sizeof(int));
        hipMemcpy(data_dense_last_d, data_dense_last, sizeof(real_type) * (*data_ptr_dense.get())[0].size());
        hipMemcpy(id_d, id, sizeof(int));

        q = std::vector<real_type>(data_ptr_dense->size() - 1); // q-Vector
        hipMemcpy(q_d, q, sizeof(real_type)*q.size(), hipMemcpyHostToDevice);
        // linear
        fmt::print("dense (linear) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = (*data_ptr_dense)[data_ptr_dense->size() - 1][(*data_ptr_dense)[0].size() - 1] * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_dense->size(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_dense->size(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_linear<<<grid_q, block_q>>>(q_d, data_dense_d, data_dense_last_d, num_rows_d, num_cols_d);
        hipDeviceSynchronize();
       
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_linear<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, data_dense_d, QA_cost_d, cost_d, num_rows_d, num_cols_d, add_d, id_d); //id = 0;
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
       
        raw_runtimes_dense_linear.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        // polynomial
        fmt::print("dense (polynomial) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = (*data_ptr_dense)[data_ptr_dense->size() - 1][(*data_ptr_dense)[0].size() - 1] * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_dense->size(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_dense->size(), 1.);
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_poly<<<grid_q, block_q>>>(q_d, data_dense_d, data_dense_last_d, num_rows_d, num_cols_d, degree_d, gamma_d, coef0_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_poly<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, data_dense_d, QA_cost_d, cost_d, num_rows_d, num_cols_d, add_d, degree_d, gamma_d, coef0_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_dense_poly.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        // radial
        fmt::print("dense (radial) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = (*data_ptr_dense)[data_ptr_dense->size() - 1][(*data_ptr_dense)[0].size() - 1] * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_dense->size(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_dense->size(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_radial<<<grid_q, block_q>>>(q_d, data_dense_d, data_dense_last_d, num_rows_d, num_cols_d, gamma_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_radial<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, data_dense_d, QA_cost_d, cost_d, num_rows_d, num_cols_d, add_d, gamma_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
       
        raw_runtimes_dense_radial.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        hipFree(q_d);
        hipFree(QA_cost_d);
        hipFree(ret_d);
        hipFree(d_d);
        hipFree(data_dense_d);
        hipFree(num_rows_d);
        hipFree(num_cols_d);
        hipFree(data_dense_last_d);
        hipFree(id_d);
    }
    
    
    // coo
    std::vector<ns> raw_runtimes_coo_linear;
    std::vector<ns> raw_runtimes_coo_poly;
    std::vector<ns> raw_runtimes_coo_radial;
    params.parse_libsvm_file_sparse(ds.path, data_ptr_coo);

    num_rows_exc_last = data_ptr_coo.get() -> get_heigth() - 1;

    range_q({ static_cast<std::size_t>(std::ceil(static_cast<real_type>(num_rows_exc_last) / static_cast<real_type>(THREAD_BLOCK_SIZE))) },
                                            { std::min<std::size_t>(THREAD_BLOCK_SIZE, num_rows_exc_last) });
    grid_q = dim3(range_q.grid[0], range_q.grid[1], range_q.grid[2]);
    block_q = dim3(range_q.block[0], range_q.block[1], range_q.block[2]); 

    grid = static_cast<std::size_t>(std::ceil(static_cast<real_type>(num_rows_exc_last) / static_cast<real_type>(boundary_size)));
    range_svm({ grid, grid }, { THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE });

    grid_svm = dim3(range_svm.grid[0], range_svm.grid[1], range_svm.grid[2]);
    block_svm = dim3(range_svm.block[0], range_svm.block[1], range_svm.block[2]); 

    for(size_t i = 0; i < cycles; i++) {
        hipMalloc((void**)&q_d, sizeof(real_type)*(data_ptr_coo -> get_height() - 1));
        hipMalloc((void**)&ret_d, sizeof(real_type)*(data_ptr_coo -> get_height()));
        hipMalloc((void**)&d_d, sizeof(real_type)*(data_ptr_coo -> get_height()));
        hipMalloc((void**)&QA_cost_d, sizeof(real_type));


        hipMalloc((void**)&nnz_coo_d, sizeof(size_t));
        hipMalloc((void**)&last_row_begin_coo_d, sizeof(size_t));
        hipMalloc((void**)&values_coo_d, sizeof(real_type)*(data_ptr_coo -> get_nnz()));
        hipMalloc((void**)&col_coo_d, sizeof(size_t)*(data_ptr_coo -> get_nnz()));
        hipMalloc((void**)&row_coo_d, sizeof(size_t)*(data_ptr_coo -> get_nnz()));

        hipMemcpy(nnz_coo_d, data_ptr_coo.get() -> get_nnz(), sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(last_row_begin_coo_d, data_ptr_coo.get() -> get_last_row_begin(), sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(values_coo_d, data_ptr_coo.get() -> get_values(), sizeof(real_type)*(data_ptr_coo -> get_nnz()), hipMemcpyHostToDevice);
        hipMemcpy(row_coo_d, data_ptr_coo.get() -> get_rows(), sizeof(real_type)*(data_ptr_coo -> get_nnz()), hipMemcpyHostToDevice);
        hipMemcpy(column_coo_d, data_ptr_coo.get() -> get_columns(), sizeof(real_type)*(data_ptr_coo -> get_nnz()), hipMemcpyHostToDevice);

        q = std::vector<real_type>(data_ptr_coo->get_height() - 1); // q-Vector
        hipMemcpy(q, q_d, sizeof(real_type)*q.size(), hipMemcpyHostToDevice);
        // linear
        fmt::print("coo (linear) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = data_ptr_coo->get_element(data_ptr_coo->get_height() - 1, data_ptr_coo->get_width() - 1) * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_coo->get_height(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_coo->get_height(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_linear<<<grid_q, block_q>>>(q_d, col_coo_d, row_coo_d, values_coo_d, last_row_begin_coo_d, nnz_coo_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_linear<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, col_coo_d, row_coo_d, values_coo_d, QA_cost_d, cost_d, nnz_coo_d, add_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_coo_linear.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        // polynomial
        fmt::print("coo (polynomial) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = data_ptr_coo->get_element(data_ptr_coo->get_height() - 1, data_ptr_coo->get_width() - 1) * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_coo->get_height(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_coo->get_height(), 1.);
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_poly<<<grid_q, block_q>>>(q_d, col_coo_d, row_coo_d, values_coo_d, last_row_begin_coo_d, nnz_coo_d, degree_d, gamma_d, coef0_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_poly<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, col_coo_d, row_coo_d, values_coo_d, QA_cost_d, cost_d, nnz_coo_d, add_d, degree_d, gamma_d, coef0_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_coo_poly.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        // radial
        fmt::print("coo (radial) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = data_ptr_coo->get_element(data_ptr_coo->get_height() - 1, data_ptr_coo->get_width() - 1) * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_coo->get_height(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_coo->get_height(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_radial<<<grid_q, block_q>>>(q_d, col_coo_d, row_coo_d, values_coo_d, last_row_begin_coo_d, nnz_coo_d, gamma_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_radial<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, col_coo_d, row_coo_d, values_coo_d, QA_cost_d, cost_d, nnz_coo_d, gamma_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_coo_radial.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        hipFree(q_d);
        hipFree(QA_cost_d);
        hipFree(ret_d);
        hipFree(d_d);

        hipFree(nnz_coo_d);
        hipFree(last_row_begin_coo_d);
        hipFree(values_coo_d);
        hipFree(col_coo_d);
        hipFree(row_coo_d);
    }
    
    // coo
    std::vector<ns> raw_runtimes_csr_linear;
    std::vector<ns> raw_runtimes_csr_poly;
    std::vector<ns> raw_runtimes_csr_radial;
    params.parse_libsvm_file_sparse(ds.path, data_ptr_csr);

    num_rows_exc_last = data_ptr_csr.get() -> get_heigth() - 1;

    range_q({ static_cast<std::size_t>(std::ceil(static_cast<real_type>(num_rows_exc_last) / static_cast<real_type>(THREAD_BLOCK_SIZE))) },
                                            { std::min<std::size_t>(THREAD_BLOCK_SIZE, num_rows_exc_last) });
    grid_q = dim3(range_q.grid[0], range_q.grid[1], range_q.grid[2]);
    block_q = dim3(range_q.block[0], range_q.block[1], range_q.block[2]); 

    grid = static_cast<std::size_t>(std::ceil(static_cast<real_type>(num_rows_exc_last) / static_cast<real_type>(boundary_size)));
    range_svm({ grid, grid }, { THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE });

    grid_svm = dim3(range_svm.grid[0], range_svm.grid[1], range_svm.grid[2]);
    block_svm = dim3(range_svm.block[0], range_svm.block[1], range_svm.block[2]);

    for(size_t i = 0; i < cycles; i++) {
        hipMalloc((void**)&q_d, sizeof(real_type)*(data_ptr_csr -> get_height()-1));
        hipMalloc((void**)&ret_d, sizeof(real_type)*(data_ptr_dense -> get_height()));
        hipMalloc((void**)&d_d, sizeof(real_type)*(data_ptr_dense -> get_height()));
        hipMalloc((void**)&QA_cost_d, sizeof(real_type));

        hipMalloc((void**)&height_csr_d, sizeof(size_t));
        hipMalloc((void**)&nnz_csr_d, sizeof(size_t));
        hipMalloc((void**)&values_csr_d, sizeof(real_type)*(data_ptr_csr -> get_nnz()));
        hipMalloc((void**)&col_csr_d, sizeof(size_t)*(data_ptr_csr -> get_nnz()));
        hipMalloc((void**)&row_csr_d, sizeof(size_t)*(data_ptr_csr -> get_height()));

        hipMemcpy(height_csr_d, data_ptr_csr.get() -> get_height(), sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(nnz_csr_d, data_ptr_csr.get() -> get_nnz(), sizeof(size_t), hipMemcpyHostToDevice);
        hipMemcpy(values_csr_d, data_ptr_csr.get() -> get_values(), sizeof(real_type)*(data_ptr_csr -> get_nnz()), hipMemcpyHostToDevice);
        hipMemcpy(row_csr_d, data_ptr_csr.get() -> get_rows(), sizeof(size_t)*(data_ptr_csr -> get_nnz()), hipMemcpyHostToDevice);
        hipMemcpy(column_csr_d, data_ptr_csr.get() -> get_columns(), sizeof(size_t)*(data_ptr_csr -> get_height()), hipMemcpyHostToDevice);

        q = std::vector<real_type>(data_ptr_csr->get_height() - 1); // q-Vector
        hipMemcpy(q, q_d, sizeof(real_type)*q.size(), hipMemcpyHostToDevice);
        // linear
        fmt::print("csr (linear) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = data_ptr_csr->get_element(data_ptr_csr->get_height() - 1, data_ptr_csr->get_width() - 1) * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_csr->get_height(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_csr->get_height(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_linear<<<grid_q, block_q>>>(q_d, col_csr_d, row_csr_d, values_csr_d, nnz_csr_d, height_csr_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_linear<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, col_csr_d, row_csr_d, values_csr_d, QA_cost_d, cost_d, nnz_csr_d, height_csr_d, add_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_csr_linear.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        // polynomial
        fmt::print("csr (polynomial) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = data_ptr_csr->get_element(data_ptr_csr->get_height() - 1, data_ptr_csr->get_width() - 1) * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_csr->get_height(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_csr->get_height(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_poly<<<grid_q, block_q>>>(q_d, col_csr_d, row_csr_d, values_csr_d, nnz_csr_d, height_csr_d, degree_d, gamma_d, coef0_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_poly<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, col_csr_d, row_csr_d, values_csr_d, QA_cost_d, cost_d, nnz_csr_d, height_csr_d, add_d, degree_d, gamma_d, coef0_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_csr_poly.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        // radial
        fmt::print("csr (radial) " + std::to_string(i + 1) + "/" + std::to_string(cycles) + " (");
        QA_cost = data_ptr_csr->get_element(data_ptr_csr->get_height() - 1, data_ptr_csr->get_width() - 1) * cost;
        hipMemcpy(QA_cost_d, QA_cost, sizeof(real_type), hipMemcpyHostToDevice);
        ret = std::vector<real_type>(data_ptr_csr->get_height(), 0.);
        hipMemcpy(ret_d, ret, sizeof(real_type)*ret.size(), hipMemcpyHostToDevice);
        d = std::vector<real_type>(data_ptr_csr->get_height(), 1.); 
        hipMemcpy(d_d, d, sizeof(real_type)*d.size(), hipMemcpyHostToDevice);

        plssvm::cuda::device_kernel_q_radial<<<grid_q, block_q>>>(q_d, col_csr_d, row_csr_d, values_csr_d, nnz_csr_d, height_csr_d, gamma_d);
        hipDeviceSynchronize();
        
        start_time = std::chrono::high_resolution_clock::now();
        plssvm::cuda::device_kernel_radial<<<grid_svm, block_svm>>>(q_d, ret_d, d_d, col_csr_d, row_csr_d, values_csr_d, QA_cost_d, cost_d, nnz_csr_d, height_csr_d, add_d, gamma_d);
        hipDeviceSynchronize();
        end_time = std::chrono::high_resolution_clock::now();
        
        raw_runtimes_csr_radial.push_back(std::chrono::round<ns>(end_time - start_time));
        fmt::print(std::to_string(std::chrono::round<ns>(end_time - start_time).count()/1000000) + "ms)\n");

        hipFree(q_d);
        hipFree(QA_cost_d);
        hipFree(ret_d);
        hipFree(d_d);

        hipFree(csr_height_d);
        hipFree(nnz_csr_d);
        hipFree(values_csr_d);
        hipFree(col_csr_d);
        hipFree(row_csr_d);
    }

    hipFree(cost_d);
    hipFree(add_d);
    hipFree(degree_d);
    hipFree(gamma_d);
    hipFree(coef0_d);
    
    sub_benchmark_names.push_back(sub_benchmark_name + "dense (linear)");
    sub_benchmark_names.push_back(sub_benchmark_name + "COO (linear)");
    sub_benchmark_names.push_back(sub_benchmark_name + "CSR (linear)");
    sub_benchmark_names.push_back(sub_benchmark_name + "dense (polynomial)");
    sub_benchmark_names.push_back(sub_benchmark_name + "COO (polynomial)");
    sub_benchmark_names.push_back(sub_benchmark_name + "CSR (polynomial)");
    sub_benchmark_names.push_back(sub_benchmark_name + "dense (radial)");
    sub_benchmark_names.push_back(sub_benchmark_name + "COO (radial)");
    sub_benchmark_names.push_back(sub_benchmark_name + "CSR (radial)");
    auto sub_benchmark_runtimes = std::vector<std::vector<ns>>{
        raw_runtimes_dense_linear,
        raw_runtimes_coo_linear,
        raw_runtimes_csr_linear,
        raw_runtimes_dense_poly,
        raw_runtimes_coo_poly,
        raw_runtimes_csr_poly,
        raw_runtimes_dense_radial,
        raw_runtimes_coo_radial,
        raw_runtimes_csr_radial};
    perform_statistics(sub_benchmark_runtimes);
}

}  // namespace plssvm::benchmarks
